#define FLT_MIN 1.175494e-38

#include <hip/hip_runtime.h>
#include <hip/hip_bf16.h>

// The compiler does not properly optimize this unless we use the CPP to make
// the constant explicit.
#define warpSize 32

extern "C" __global__ void fp32_to_bf16(const float* input, __hip_bfloat16* output, int size) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < size) {
        output[i] = __float2bfloat16_rz(input[i]);
    }
}

extern "C" __global__ void batched_softmax(float *data, int size, int batch_size) {
    extern __shared__ float shared[];
    unsigned int tid = threadIdx.x;
    unsigned int warpId = threadIdx.x / warpSize;
    unsigned int laneId = threadIdx.x % warpSize;

    int batch_id = blockIdx.x; // Simplified batch ID, assumes one block per batch

    if (batch_id >= batch_size) return;

    float* x = data + batch_id * size;

    // ----- 1. Partial Max Value Calculation -----
    float partial_max = FLT_MIN; // Initialize to minimum float
    for (int i = tid; i < size; i += blockDim.x) {
        partial_max = fmaxf(x[i], partial_max);
    }

    // ----- 2. Warp Reduction for Max -----
    float warpMax = partial_max;
    for (int offset = warpSize / 2; offset > 0; offset >>= 1) {
        float otherValue = __shfl_down_sync(0xFFFFFFFF, warpMax, offset);
        warpMax = fmaxf(warpMax, otherValue);
    }

    // ----- 3. Warp Leader Writes to Shared Memory -----
    if (laneId == 0) {
        shared[warpId] = warpMax;
    }
    __syncthreads();

    // ----- 4. Block Reduction for Max -----
    float blockMax = shared[laneId];
    for (int offset = warpSize / 2; offset > 0; offset >>= 1) {
      float otherValue = __shfl_down_sync(0xFFFFFFFF, blockMax, offset);
      blockMax = fmaxf(blockMax, otherValue);
    }

    // ----- 5. Broadcast the block max -----
    float maxVal = __shfl_sync(0xFFFFFFFF, blockMax, 0);

    // ----- 6. Calculate Partial Exp and Sum -----
    float partial_sum = 0.0f;
    for (int i = tid; i < size; i += blockDim.x) {
        x[i] = __expf(x[i] - maxVal);
        partial_sum += x[i];
    }

    // ----- 7. Warp Reduction for Sum -----
    float warpSum = partial_sum;
    for (int offset = warpSize / 2; offset > 0; offset >>= 1) {
        float otherValue = __shfl_down_sync(0xFFFFFFFF, warpSum, offset);
        warpSum += otherValue;
    }

    // ----- 8. Warp Leader Writes Sum to Shared Memory -----
    if (laneId == 0) {
        shared[warpId] = warpSum;
    }
    __syncthreads();

    // ----- 9. Block Reduction for Sum -----
    float blockSum = shared[laneId];
    for (int offset = warpSize / 2; offset > 0; offset >>= 1) {
      float otherValue = __shfl_down_sync(0xFFFFFFFF, blockSum, offset);
      blockSum += otherValue;
    }

    // ----- 10. Broadcast the block Sum -----
    float sum = __shfl_sync(0xFFFFFFFF, blockSum, 0);

    // ----- 11. Normalize each element -----
    float rsum = 1.0f / sum;
    for (int i = tid; i < size; i += blockDim.x) {
        x[i] *= rsum;
    }
}

extern "C" __global__ void swiGLU(float* hb, float* hb2, int hidden_dim) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < hidden_dim) {
        float val = hb[i];
        // silu(x) = x * sigmoid(x)
        val *= 1.0f / (1.0f + __expf(-val));  //More efficient sigmoid calculation
        val *= hb2[i];
        hb[i] = val;
    }
}

extern "C" __global__ void rope_rotary_encoding(
    float* __restrict__ q,        // Query matrix
    float* __restrict__ k,        // Key matrix
    const int n_heads,            // Number of attention heads
    const int n_kv_heads,         // Number of key-value heads
    const int head_size,          // Size of each head (assumed to be even)
    const int pos)                // Position index
{
    // Each thread computes a specific (head, j) pair
    int head = blockIdx.x;          // Head index (i)
    int j = threadIdx.x * 2;        // Position within head (stride of 2 for complex pairs)

    if (head >= n_heads || j >= head_size)
        return;

    // Compute the frequency for this position
    float freq = __powf(500000.0f, (float)-j / (float)head_size);
    float val = pos * freq;
    float fcr = __cosf(val);
    float fci = __sinf(val);

    // Load q values (q0 and q1)
    float q0 = q[head * head_size + j];
    float q1 = q[head * head_size + j + 1];

    // Rotate q values
    q[head * head_size + j]     = q0 * fcr - q1 * fci;
    q[head * head_size + j + 1] = q0 * fci + q1 * fcr;

    // Rotate k values only if within n_kv_heads
    if (head < n_kv_heads) {
        float k0 = k[head * head_size + j];
        float k1 = k[head * head_size + j + 1];

        k[head * head_size + j]     = k0 * fcr - k1 * fci;
        k[head * head_size + j + 1] = k0 * fci + k1 * fcr;
    }
}


extern "C" __global__ void rmsnorm(float *o, float *x, float *weight, int size) {
    unsigned int tid = threadIdx.x;
    unsigned int warpId = threadIdx.x / warpSize;
    unsigned int laneId = threadIdx.x % warpSize;

    // ----- 1. Calculate partial sum of squares -----
    float partial_ss = 0.0f;
    for (int j = tid; j < size; j += blockDim.x) {
        partial_ss += x[j] * x[j];
    }

    // ----- 2. Warp Reduction (using shuffles) -----
    float warpSum = partial_ss;
    for (int offset = warpSize / 2; offset > 0; offset >>= 1) {
      float otherValue = __shfl_down_sync(0xFFFFFFFF, warpSum, offset);
      warpSum += otherValue;
    }

    // ----- 3. Warp Leader Writes to Shared Memory -----
    extern __shared__ float shared[];
    if (laneId == 0) {
      shared[warpId] = warpSum;
    }
    __syncthreads();

    // ----- 4. Block Reduction (using shuffles) -----
    float blockSum = shared[laneId];
    for (int offset = warpSize / 2; offset > 0; offset >>= 1)
      blockSum += __shfl_down_sync(0xffffffff, blockSum, offset);

    // ----- 5. Broadcast the block sum (all threads do it)-----
    blockSum = __shfl_sync(0xFFFFFFFF, blockSum, 0);

    // ----- 6. Calculate global RMS normalization factor (all threads calculate it) -----
    float ss = blockSum;
    ss /= size;
    ss += 1e-5f;
    ss = rsqrtf(ss);

    // ----- 7. Normalize and scale each element -----
    for (int j = tid; j < size; j += blockDim.x) {
      o[j] = weight[j] * (ss * x[j]);
    }
}
